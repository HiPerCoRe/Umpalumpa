#include "hip/hip_runtime.h"
#include <starpu.h>

static __global__ void vector_mult_cuda(unsigned n, float* val, float factor) {
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) val[i] *= factor;
  if (i == 2) printf("hello from cuda\n");
}

extern "C" void scal_cuda_func(void* buffers[], void* _args) {
  float* factor = (float*)_args; /*length of the vector*/
  unsigned n =
      STARPU_VECTOR_GET_NX(buffers[0]); /*local copy of the vector pointer*/
  float* val = (float*)STARPU_VECTOR_GET_PTR(buffers[0]);
  unsigned threads_per_block = 64;
  unsigned nblocks = (n + threads_per_block - 1) / threads_per_block;
  vector_mult_cuda<<<nblocks, threads_per_block, 0,
                     starpu_cuda_get_local_stream()>>>(n, val, *factor);
  hipError_t status = hipGetLastError();
  if (status != hipSuccess) STARPU_CUDA_REPORT_ERROR(status);
  hipStreamSynchronize(starpu_cuda_get_local_stream());
}
