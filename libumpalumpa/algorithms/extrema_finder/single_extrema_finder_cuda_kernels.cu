#include "hip/hip_runtime.h"
#pragma once
#include <libumpalumpa/data/size.hpp>

#ifndef blockSizeX
#define blockSizeX 1
#endif

#ifndef blockSizeY
#define blockSizeY 1
#endif

#ifndef blockSize
#define blockSize (blockSizeX * blockSizeY)
#endif

template<typename T, typename T2, typename C>
__device__ void update(
  const C &comp,
  T2 &orig,
  const T *__restrict data,
  unsigned index)
{
  T tmp = data[index];
  if (comp(tmp, index, orig.x, orig.y)) {
    orig.x = tmp;
    orig.y = (T)index;
  }
}

template<typename T, typename C>
__device__ T update(
  const C &comp,
  T &orig,
  T &cand)
{
  if (comp(cand.x, cand.y, orig.x, orig.y)) {
    orig.x = cand.x;
    orig.y = cand.y;
  }
  return orig;
}

template<typename T, typename C>
__device__ void findUniversalInSharedMem(
  const C &comp,
  T &ldata,
  unsigned int tid)
{
  // we have read all data, one of the thread knows the result
  __shared__ T sdata[blockSize];
  sdata[tid] = ldata;
  __syncthreads();// wait till all threads store their data
  // reduce
#pragma unroll
  for (auto counter = blockSize / 2; counter >= 32; counter /= 2) {
    if (tid < counter) {
      sdata[tid] = update(
        comp,
        ldata,
        sdata[tid + counter]);
    }
    __syncthreads();
  }
  // manually unwrap last warp for better performance
  // many of these blocks will be optimized out by compiler based on template
  if ((blockSize >= 32) && (tid < 16)) {
    sdata[tid] = update(
      comp,
      ldata,
      sdata[tid + 16]);
  }
  __syncthreads();
  if ((blockSize >= 16) && (tid < 8)) {
    sdata[tid] = update(
      comp,
      ldata,
      sdata[tid + 8]);
  }
  __syncthreads();
  if ((blockSize >= 8) && (tid < 4)) {
    sdata[tid] = update(
      comp,
      ldata,
      sdata[tid + 4]);
  }
  __syncthreads();
  if ((blockSize >= 4) && (tid < 2)) {
    sdata[tid] = update(
      comp,
      ldata,
      sdata[tid + 2]);
  }
  __syncthreads();
  if ((blockSize >= 2) && (tid < 1)) {
    sdata[tid] = update(
      comp,
      ldata,
      sdata[tid + 1]);
  }
  __syncthreads();
}

// template <typename T, typename T2, unsigned blockSize, typename C>
// __device__
// void findMax1D(
//         const C &comp,
//         T startVal,
//         const T * __restrict__ in,
//         float * __restrict__ outPos,
//         T * __restrict__ outVal,
//         unsigned samples)
// {
__global__ void findMax(
  // const C &comp,
  // T startVal,
  float * __restrict__ in,
  float * __restrict__ outVal,
  float * __restrict__ outPos,
  umpalumpa::data::Size size)
{
  // return true IFF first value is bigger than the second value, or they are the same and
  // the positio of the first value is lower -> returns the biggers value at the lowest position
  auto comp = [] (float l, unsigned li, float r, unsigned ri) { return l > r || (l == r && li < ri); };
  // one block processes one signal
  // map each thread to some sample of the signal
  // blockSize == noOfThreads
  auto tid = threadIdx.x;
  auto signal = blockIdx.x;
  
  // load data from global memory
  if (tid >= size.single) return;
  auto samples = size.single;
  const float *data = in + (signal * samples);

  float2 ldata;
  ldata.x = data[tid];
  ldata.y = tid;
  for (auto i = tid + blockSize; i < samples; i += blockSize) {
    update(
      comp,
      ldata,
      data,
      i);
  }
  __syncthreads();// wait till all threads are ready
  findUniversalInSharedMem(//<float2, blockSize>(
    comp,
    ldata,
    tid);

  // last thread now holds the result
  if (tid == 0) {
    if (nullptr != outVal) { outVal[signal] = ldata.x; }
    if (nullptr != outPos) { 
      const auto dim = size.GetDimAsNumber();
      const unsigned location = ldata.y;
      auto *dest = outPos + signal * dim;
      if (1 == dim) {
        dest[0] = static_cast<float>(location);
      } else if (2 == dim) {
        auto y = location / size.x;
        auto x = location % size.x;
        dest[0] = x;
        dest[1] = y;
      } else if (3 == dim) {
        size_t z = location / (size.x * size.y);
        size_t tmp = location % (size.x * size.y);
        size_t y = tmp / size.x;
        size_t x = tmp % size.x;
        dest[0] = static_cast<float>(x);
        dest[1] = static_cast<float>(y);
        dest[2] = static_cast<float>(z);
      } else {
        dest[signal] = nanf("");
      }
    }
  }
}

/**
 * Find sub-pixel location or value of the extrema.
 * Data has to contain at least one (1) value.
 * Returned location is calculated by relative weigting in the given
 * window using the value contribution. Should the window reach behind the boundaries, those
 * values will be ignored. Only odd sizes of the window are valid.
 *
 * All checks are expected to be done by caller
 **/
 template<typename T, unsigned WINDOW>
 __global__ 
 void RefineLocation(float *__restrict__ locs,
   T *const __restrict__ data,
   const umpalumpa::data::Size size)
{
  // map one thread per signal
  auto n = threadIdx.x;
  if (n >= size.single) return;
  using umpalumpa::data::Dimensionality;
  auto half = (WINDOW - 1) / 2;
  const auto dim = size.GetDimAsNumber();
  if ((dim > 0) && (dim <= 3)) {
      auto *ptrLoc = locs + n * size.GetDimAsNumber();
      auto *ptr = data + n * size.single;
      auto refX = static_cast<size_t>(ptrLoc[0]);
      auto refY = (size.GetDimAsNumber() > 1) ? static_cast<size_t>(ptrLoc[1]) : 0;
      auto refZ = (size.GetDimAsNumber() > 2) ? static_cast<size_t>(ptrLoc[2]) : 0;
      auto refVal = data[n * size.single + refZ * size.x * size.y + refY * size.x + refX];
      // careful with unsigned operations
      auto startX = (half > refX) ? 0 : refX - half;
      auto endX = min(half + refX, size.x - 1);
      auto startY = (half > refY) ? 0 : refY - half;
      auto endY = min(half + refY, size.y - 1);
      auto startZ = (half > refZ) ? 0 : refZ - half;
      auto endZ = min(half + refZ, size.z - 1);
      float sumLocX = 0;
      float sumLocY = 0;
      float sumLocZ = 0;
      float sumWeight = 0;
      for (auto z = startZ; z <= endZ; ++z) {
        for (auto y = startY; y <= endY; ++y) {
          for (auto x = startX; x <= endX; ++x) {
            auto i = z * size.x * size.y + y * size.x + x;
            auto relVal = ptr[i] / refVal;
            sumWeight += relVal;
            sumLocX += static_cast<float>(x) * relVal;
            sumLocY += static_cast<float>(y) * relVal;
            sumLocZ += static_cast<float>(z) * relVal;
          }
        }
      }
      ptrLoc[0] = sumLocX / sumWeight;
      if (size.GetDimAsNumber() > 1) { ptrLoc[1] = sumLocY / sumWeight; }
      if (size.GetDimAsNumber() > 2) { ptrLoc[2] = sumLocZ / sumWeight; }
    return;
  }
  // otherwise we don't know what to do, so 'report' it
  for (size_t n = 0; n < size.n * size.GetDimAsNumber(); ++n) {
    locs[n] = nanf("");
  }
}

template<typename T>
__global__ void findMaxRect(T *in,
  umpalumpa::data::Size inSize,
  T *outVal,
  T *outPos,
  unsigned offsetX,
  unsigned offsetY,
  unsigned rectWidth,
  unsigned rectHeight)
{
  unsigned signal = blockIdx.x;

  const float *data = in + (signal * inSize.single);

  // return true IFF first value is bigger than the second value, or they are the same and
  // the positio of the first value is lower -> returns the biggers value at the lowest position
  auto comp = [] (float l, unsigned li, float r, unsigned ri) { return l > r || (l == r && li < ri); };

  float2 ldata;
  ldata.x = data[offsetY * inSize.x + offsetX];
  ldata.y = -1;
  for (unsigned tIdy = offsetY + threadIdx.y; tIdy < offsetY + rectHeight; tIdy += blockSizeY) {
    for (unsigned tIdx = offsetX + threadIdx.x; tIdx < offsetX + rectWidth; tIdx += blockSizeX) {
      update(comp, ldata, data, tIdy * inSize.x + tIdx);
    }
  }

  __syncthreads();// wait till all threads are ready
  findUniversalInSharedMem(//<float2, blockSize>(
    comp,
    ldata,
    threadIdx.y * blockSizeX + threadIdx.x);

  // last thread now holds the result
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    if (nullptr != outVal) { outVal[signal] = ldata.x; }
    if (nullptr != outPos) { outPos[signal] = ldata.y; }
  }
}

// template <typename T, unsigned blockSize, typename C>
// __global__
// void findUniversal(
//         const C &comp,
//         T startVal,
//         const T * __restrict__ in,
//         float * __restrict__ outPos,
//         T * __restrict__ outVal,
//         unsigned samples)
// {
//     if (std::is_same<T, float> ::value) {
//         findMax1D<float, float2, blockSize> (
//                 comp,
//                 startVal,
//                 (float*)in,
//                 outPos,
//                 (float*)outVal,
//                 samples);
//     } else if (std::is_same<T, double> ::value) {
//         findMax1D<double, double2, blockSize>(
//                 comp,
//                 startVal,
//                 (double*)in,
//                 outPos,
//                 (double*)outVal,
//                 samples);
//     }
// }
